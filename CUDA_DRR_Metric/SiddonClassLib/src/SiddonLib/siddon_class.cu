#include "hip/hip_runtime.h"
#include <cstdio>
#include "siddon_class.cuh"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""


__device__ const float epsilon = 2.22045e-016; // to compare double float values

// auxiliary functions

__device__ void get_dest(int idx, float *dest_array, float *dest) {

	dest[0] = dest_array[0 + 3 * idx];
	dest[1] = dest_array[1 + 3 * idx];
	dest[2] = dest_array[2 + 3 * idx];

}


__device__ void compute_alpha_x(const float &X0,
	const float &spacing_x,
	const int &i,
	const float &source_x,
	const float &dest_x,
	float &alpha_x) {

	alpha_x = ((X0 + static_cast<float>(i)*spacing_x) - source_x) / (dest_x - source_x);

}


__device__ void compute_alpha_y(const float &Y0,
	const float &spacing_y,
	const int &j,
	const float &source_y,
	const float &dest_y,
	float &alpha_y) {

	alpha_y = ((Y0 + static_cast<float>(j)*spacing_y) - source_y) / (dest_y - source_y);
}


__device__ void compute_alpha_z(const float &Z0,
	const float &spacing_z,
	const int &k,
	const float &source_z,
	const float &dest_z,
	float &alpha_z) {

	alpha_z = ((Z0 + static_cast<float>(k)*spacing_z) - source_z) / (dest_z - source_z);
}


__device__ void compute_phi_x(const float &X0,
	const float &spacing_x,
	float &alpha,
	const float &source_x,
	const float &dest_x,
	float &phi_x) {

	phi_x = (source_x + alpha*(dest_x - source_x) - X0) / spacing_x;
}


__device__ void compute_phi_y(const float &Y0,
	const float &spacing_y,
	float &alpha,
	const float &source_y,
	const float &dest_y,
	float &phi_y) {

	phi_y = (source_y + alpha*(dest_y - source_y) - Y0) / spacing_y;
}


__device__ void compute_phi_z(const float &Z0,
	const float &spacing_z,
	float &alpha,
	const float &source_z,
	const float &dest_z,
	float &phi_z) {

	phi_z = (source_z + alpha*(dest_z - source_z) - Z0) / spacing_z;
}

__device__ void update_idx(unsigned int &i_v, unsigned int &j_v, unsigned int &k_v, const int &size_x, const int &size_y, int &arrayIdx) {

	arrayIdx = i_v + size_x * (j_v + size_y * k_v);
}


__global__ void generateDRR_kernel(float *DRRarray,
	float *source,
	float *DestArray,
	int DRRsize0,
	int DRRsize1,
	float *movImgArray,
	int *MovSize,
	float *MovSpacing,
	float X0, float Y0, float Z0) {

	// DRR image indeces
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	// DRR array index
	int DRRidx = row + DRRsize0 * col;

	// moving image total size
	int totalMovSize = MovSize[0] * MovSize[1] * MovSize[2];

	// printf("Thread index %i\n", DRRidx);
	// printf("row, col, row * col: %d, %d, %d\n", DRRsize0, DRRsize1, DRRsize0 * DRRsize1);

	if (DRRidx < DRRsize0 * DRRsize1) { // checks if thread index is within the length of the DRR array

		// --- declaration of variables for Siddon ---
		float alpha_min, alpha_max;
		float alpha_x_min, alpha_x_max, alpha_y_min, alpha_y_max, alpha_z_min, alpha_z_max;
		int i_min, i_max, j_min, j_max, k_min, k_max; // indeces corresponding to first and last intersected voxels
		float alpha_current;
		float alpha_x_next;
		float alpha_y_next;
		float alpha_z_next;
		float distance;
		int arrayIdx;
		int arrayIdx_old;
		unsigned int i_v, j_v, k_v;
		float alpha_first_pixel;
		float density_value = 0.;
		
		// --- initialize DRR's density value ---
		DRRarray[DRRidx] = 0;

		// --- define destination point based on DRR array index --- 
		float dest[3];
		get_dest(DRRidx, DestArray, dest);

		// --- source-to-destination distance --- 
		distance = sqrtf((dest[0] - source[0])*(dest[0] - source[0]) +
			(dest[1] - source[1])*(dest[1] - source[1]) +
			(dest[2] - source[2])*(dest[2] - source[2]));

		float dx = MovSpacing[0] / fabsf(dest[0] - source[0]);
		float dy = MovSpacing[1] / fabsf(dest[1] - source[1]);
		float dz = MovSpacing[2] / fabsf(dest[2] - source[2]);

		// --- find alpha_min and alpha_max
		// initialize alpha_min and alpha_max
		alpha_min = 0.;
		alpha_max = 1.;

		// X
		if (fabsf(dest[0] - source[0]) > epsilon) {

			float alpha_x0 = (X0 - source[0]) / (dest[0] - source[0]);
			float alpha_xN;
			compute_alpha_x(X0, MovSpacing[0], MovSize[0], source[0], dest[0], alpha_xN);
			alpha_x_min = fminf(alpha_x0, alpha_xN);
			alpha_x_max = fmaxf(alpha_x0, alpha_xN);
			if (alpha_x_min > alpha_min) { alpha_min = alpha_x_min; };
			if (alpha_x_max < alpha_max) { alpha_max = alpha_x_max; };
			//printf("alpha_min: %f\n", alpha_min);
			//printf("alpha_max: %f\n", alpha_max);

		}

		// Y
		if (fabsf(dest[1] - source[1]) > epsilon) {

			float alpha_y0 = (Y0 - source[1]) / (dest[1] - source[1]);
			float alpha_yN;
			compute_alpha_y(Y0, MovSpacing[1], MovSize[1], source[1], dest[1], alpha_yN);
			alpha_y_min = fminf(alpha_y0, alpha_yN);
			alpha_y_max = fmaxf(alpha_y0, alpha_yN);
			if (alpha_y_min > alpha_min) { alpha_min = alpha_y_min; };
			if (alpha_y_max < alpha_max) { alpha_max = alpha_y_max; };

		}

		// Z
		if (fabsf(dest[2] - source[2]) > epsilon) {

			float alpha_z0 = (Z0 - source[2]) / (dest[2] - source[2]);
			float alpha_zN;
			compute_alpha_z(Z0, MovSpacing[2], MovSize[2], source[2], dest[2], alpha_zN);
			alpha_z_min = fminf(alpha_z0, alpha_zN);
			alpha_z_max = fmaxf(alpha_z0, alpha_zN);
			if (alpha_z_min > alpha_min) { alpha_min = alpha_z_min; };
			if (alpha_z_max < alpha_max) { alpha_max = alpha_z_max; };

		}

		// if (DRRidx == 0){
		// printf("Alpha min = %f\n", alpha_min);
		// printf("Alpha max = %f\n", alpha_max);
		// printf("dx = %f\n", dx);
		// printf("dy = %f\n", dy);
		// printf("dz = %f\n", dz);
		// printf("distance = %f\n", distance);
		// }

		// --- initialize alpha --- 
		alpha_current = alpha_min;

		if (alpha_min < alpha_max) {

			// compute i_min, i_max and initialize alpha_x_next 
			if (dest[0] - source[0] > 0.) {

				// i_min
				if (fabsf(alpha_min - alpha_x_min) < epsilon) { i_min = 1; } // in other words: if source point is out of 3D model(x plane parallel region)
				else {
					float phi_x;
					compute_phi_x(X0, MovSpacing[0], alpha_min, source[0], dest[0], phi_x);
					i_min = ceil(phi_x);
				}

				// i_max
				if (fabsf(alpha_max - alpha_x_max) < epsilon) { i_max = MovSize[0] - 1; } // in other words: if destination point is out of 3D model(x plane parallel region)
				else {
					float phi_x;
					compute_phi_x(X0, MovSpacing[0], alpha_max, source[0], dest[0], phi_x);
					i_max = floor(phi_x);
				}

				// initialize alpha_x_next
				compute_alpha_x(X0, MovSpacing[0], i_min, source[0], dest[0], alpha_x_next);
			}
			else {

				// i_max
				if (fabsf(alpha_min - alpha_x_min) < epsilon) { i_max = MovSize[0] - 1; }
				else {
					float phi_x;
					compute_phi_x(X0, MovSpacing[0], alpha_min, source[0], dest[0], phi_x);
					i_max = floor(phi_x);
				}

				// i_min
				if (fabsf(alpha_max - alpha_x_max) < epsilon) { i_min = 1; }
				else {
					float phi_x;
					compute_phi_x(X0, MovSpacing[0], alpha_max, source[0], dest[0], phi_x);
					i_min = ceil(phi_x);
				}

				// initialize alpha_x_next
				compute_alpha_x(X0, MovSpacing[0], i_max, source[0], dest[0], alpha_x_next);
			}

			// compute j_min, j_max and initialize alpha_y_next 
			if (dest[1] - source[1] > 0.) {

				// j_min
				if (fabsf(alpha_min - alpha_y_min) < epsilon) { j_min = 1; }
				else {
					float phi_y;
					compute_phi_y(Y0, MovSpacing[1], alpha_min, source[1], dest[1], phi_y);
					//printf("phi_y: %f\n", phi_y);
					j_min = ceil(phi_y);
					//printf("j_min: %f\n", j_min);
				}

				// j_max
				if (fabsf(alpha_max - alpha_y_max) < epsilon) { j_max = MovSize[1] - 1; }
				else {
					float phi_y;
					compute_phi_y(Y0, MovSpacing[1], alpha_max, source[1], dest[1], phi_y);
					j_max = floor(phi_y);
				}

				// initialize alpha_y_next
				compute_alpha_y(Y0, MovSpacing[1], j_min, source[1], dest[1], alpha_y_next);
			}
			else {

				// j_max
				if (fabsf(alpha_min - alpha_y_min) < epsilon) { j_max = MovSize[1] - 1; }
				else {
					float phi_y;
					compute_phi_y(Y0, MovSpacing[1], alpha_min, source[1], dest[1], phi_y);
					j_max = floor(phi_y);
				}

				// j_min
				if (fabsf(alpha_max - alpha_y_max) < epsilon) { j_min = 1; }
				else {
					float phi_y;
					compute_phi_y(Y0, MovSpacing[1], alpha_max, source[1], dest[1], phi_y);
					j_min = ceil(phi_y);
				}

				// initialize alpha_y_next
				compute_alpha_y(Y0, MovSpacing[1], j_max, source[1], dest[1], alpha_y_next);
			}

			// compute k_min, k_max and initialize alpha_z_next 
			if (dest[2] - source[2] > 0.) {

				// k_min
				if (fabsf(alpha_min - alpha_z_min) < epsilon) { k_min = 1; }
				else {
					float phi_z;
					compute_phi_z(Z0, MovSpacing[2], alpha_min, source[2], dest[2], phi_z);
					k_min = ceil(phi_z);
				}

				// k_max
				if (fabsf(alpha_max - alpha_z_max) < epsilon) { k_max = MovSize[2] - 1; }
				else {
					float phi_z;
					compute_phi_z(Z0, MovSpacing[2], alpha_max, source[2], dest[2], phi_z);
					k_max = floor(phi_z);
				}

				// initialize alpha_z_next
				compute_alpha_z(Z0, MovSpacing[2], k_min, source[2], dest[2], alpha_z_next);
			}
			else {

				// k_max
				if (fabsf(alpha_min - alpha_z_min) < epsilon) { k_max = MovSize[2] - 1; }
				else {
					float phi_z;
					compute_phi_z(Z0, MovSpacing[2], alpha_min, source[2], dest[2], phi_z);
					k_max = floor(phi_z);
				}

				// k_min
				if (fabsf(alpha_max - alpha_z_max) < epsilon) { k_min = 1; }
				else {
					float phi_z;
					compute_phi_z(Z0, MovSpacing[2], alpha_max, source[2], dest[2], phi_z);
					k_min = ceil(phi_z);
				}

				// initialize alpha_z_next
				compute_alpha_z(Z0, MovSpacing[2], k_max, source[2], dest[2], alpha_z_next);
			}

			// if (DRRidx == 0) {
			// 	printf("i_min, i_max, Alpha_x_next = %d %d %f\n", i_min, i_max, alpha_x_next);
			// 	printf("j_min, j_max, Alpha_y_next = %d %d %f\n", j_min, j_max, alpha_y_next);
			// 	printf("k_min, k_max, Alpha_z_next = %d %d %f\n", k_min, k_max, alpha_z_next);
			// }

			// --- initialize first intersected pixel i_v, j_v, k_v --- 
			if ((alpha_y_next < alpha_x_next) && (alpha_y_next < alpha_z_next)) {

				alpha_first_pixel = (alpha_y_next + alpha_min) / 2.;
			}
			else if (alpha_x_next < alpha_z_next) {

				alpha_first_pixel = (alpha_x_next + alpha_min) / 2.;
			}
			else {

				alpha_first_pixel = (alpha_z_next + alpha_min) / 2.;
			}


			float phi_x = 0.;
			float phi_y = 0.;
			float phi_z = 0.;
			compute_phi_x(X0, MovSpacing[0], alpha_first_pixel, source[0], dest[0], phi_x);
			i_v = floor(phi_x);
			compute_phi_y(Y0, MovSpacing[1], alpha_first_pixel, source[1], dest[1], phi_y);
			j_v = floor(phi_y);
			compute_phi_z(Z0, MovSpacing[2], alpha_first_pixel, source[2], dest[2], phi_z);
			k_v = floor(phi_z);

			// initialize array index of first intersected pixel
			arrayIdx = i_v + MovSize[0] * (j_v + MovSize[1] * k_v);
			arrayIdx_old = i_v + MovSize[0] * (j_v + MovSize[1] * k_v);
			
			//printf("pixel: %f\n", movImgArray[50 + MovSize[0] * (40 + MovSize[1] * 60)]);

			// if (DRRidx == 0) {
			// 	printf("i_v, j_v, k_v = %d %d %d\n", i_v, j_v, k_v);
			// 	printf("arrayIdx, arrayIdx_old = %d %d\n", arrayIdx, arrayIdx_old);
			// }

			// iterator indeces
			int stop = (i_max - i_min + 1) + (j_max - j_min + 1) + (k_max - k_min + 1);
			int iter = 0;

			//while (alpha_current < 1. && alpha_current < alpha_max) {
			while (iter < stop) {

				float l;

				// next intersection plane is y
				if ((alpha_y_next < alpha_x_next) && (alpha_y_next < alpha_z_next)) {

					//T alpha_mid = (alpha_current + alpha_y_next) / 2.;
					l = (alpha_y_next - alpha_current);

					alpha_current = alpha_y_next;

					// update
					alpha_y_next += dy;
					j_v += (dest[1] - source[1] > 0.) ? 1 : -1;

				}

				else if (alpha_x_next < alpha_z_next) {

					// next intersection plane is x
					//T alpha_mid = (alpha_current + alpha_x_next) / 2.;
					l = (alpha_x_next - alpha_current);

					alpha_current = alpha_x_next;

					// update
					alpha_x_next += dx;
					i_v += (dest[0] - source[0] > 0.) ? 1 : -1;


				}

				else {

					// next intersection plane is z
					//T alpha_mid = (alpha_current + alpha_z_next) / 2.;
					l = (alpha_z_next - alpha_current);

					alpha_current = alpha_z_next;

					// update
					alpha_z_next += dz;
					k_v += (dest[2] - source[2] > 0.) ? 1 : -1;
				}

				// update array index
				update_idx(i_v, j_v, k_v, MovSize[0], MovSize[1], arrayIdx);

				//if (arrayIdx < 0) {
				//	printf("arrayIdx negative! %i", arrayIdx);
				//}

				// if (arrayIdx > MovSize[0] * MovSize[1] * MovSize[2]){
				// 	printf("OUT OF BOUND!!!");
				// 	printf("arrayIdx:%d", arrayIdx);
				// }

				// if (DRRidx == 0){
				// 	printf("arrayIdx:%d\n", arrayIdx);
				// }

				if (0 < arrayIdx_old && arrayIdx_old < totalMovSize){
				// update density value
					if ((movImgArray[arrayIdx_old]) > 0.) { // Note: don't use fabsf(movImgArray[arrayIdx_old]) > epsilon !!!

						density_value += (movImgArray[arrayIdx_old]) * l;
						
						// if(DRRidx == 134885)
						// {
						// 	printf("l: %f\n", l);
						// 	printf("movImgArray[arrayIdx_old]: %f\n", movImgArray[arrayIdx_old]);
						// 	printf("density_value: %f\n", density_value);
						// }
						
						}
				}

				// update arrayIdx
				arrayIdx_old = arrayIdx;

				// update iter
				iter += 1;


			}

			// multiply by the distance
			density_value *= distance;
			//if(DRRidx == 134885)
			//{
			//	printf("final density value: %f\n", density_value);
			//}
			//std::cout << density_value << std::endl;

		}

		// update density value array
		// if (DRRidx < 0){
		// 	printf("OUT OF BOUND!!!");
		// }

		DRRarray[DRRidx] = density_value;
		

		// if(DRRidx == 0)
		// {
		// 	printf("finalfinal density value: %f\n", DRRarray[DRRidx]);
		// }

	}

}

__global__ void computeMetric_kernel(float *weightSum,
	int *weightNum,
	int DRRsize0,
	int DRRsize1,
	float *DRRArray,
	float FixedThreshold,
	float DRRThreshold,
	float *FixedArray) {

	// DRR and fixed image indeces
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	// DRR and fixed array index
	int idx = row + DRRsize0 * col;

	// printf("Thread index %i\n", DRRidx);
	// printf("row, col: %d, %d\n", row, col);

	const int s_idx = threadIdx.y * blockDim.x + threadIdx.x;
	
	// allocate shared memory in each block
	__shared__ float s_sum[256];
	__shared__ int s_num[256];

	s_sum[s_idx] = 0;
	s_num[s_idx] = 0;

	if (DRRsize0 < idx && idx < DRRsize0 * (DRRsize1 - 1) && 
		idx % DRRsize0 != 0 && 
		idx % DRRsize0 != DRRsize0 - 1
		) { // checks if thread index is within the length of the DRR array

		// printf("index:%d\n", idx);

		float drrPixel = DRRArray[idx];
		float drrPixel_left = DRRArray[idx - 1];
		float drrPixel_right = DRRArray[idx + 1];
		float drrPixel_top = DRRArray[idx - DRRsize0];
		float drrPixel_bottom = DRRArray[idx + DRRsize0];
		float drrPixel_left_top = DRRArray[idx - DRRsize0 - 1];
		float drrPixel_right_top = DRRArray[idx - DRRsize0 + 1];
		float drrPixel_left_bottom = DRRArray[idx + DRRsize0 - 1];
		float drrPixel_right_bottom = DRRArray[idx + DRRsize0 + 1];
		float fixedPixel = FixedArray[idx];
		float fixedPixel_left = FixedArray[idx - 1];
		float fixedPixel_right = FixedArray[idx + 1];
		float fixedPixel_top = FixedArray[idx - DRRsize0];
		float fixedPixel_bottom = FixedArray[idx + DRRsize0];
		float fixedPixel_left_top = FixedArray[idx - DRRsize0 - 1];
		float fixedPixel_right_top = FixedArray[idx - DRRsize0 + 1];
		float fixedPixel_left_bottom = FixedArray[idx + DRRsize0 - 1];
		float fixedPixel_right_bottom = FixedArray[idx + DRRsize0 + 1];

		// if(idx == 599) {
		// 	printf("drrPixel:%f\n", drrPixel);
		// 	printf("drrPixel:%f\n", drrPixel_left);
		// 	printf("drrPixel:%f\n", drrPixel_right);
		// 	printf("drrPixel:%f\n", drrPixel_top);
		// 	printf("drrPixel:%f\n", drrPixel_bottom);
		// 	printf("fixedPixel:%f\n", fixedPixel);
		// 	printf("fixedPixel:%f\n", fixedPixel_left);
		// 	printf("fixedPixel:%f\n", fixedPixel_right);
		// 	printf("fixedPixel:%f\n", fixedPixel_top);
		// 	printf("fixedPixel:%f\n", fixedPixel_bottom);
		// }

		float m_dx = (drrPixel_right_top + 2 * drrPixel_right + drrPixel_right_bottom) - (drrPixel_left_top + 2 * drrPixel_left + drrPixel_left_bottom);
		float m_dy = (drrPixel_left_bottom + 2 * drrPixel_bottom + drrPixel_right_bottom) - (drrPixel_left_top + 2 * drrPixel_top + drrPixel_right_top);
		float f_dx = (fixedPixel_right_top + 2 * fixedPixel_right + fixedPixel_right_bottom) - (fixedPixel_left_top + 2 * fixedPixel_left + fixedPixel_left_bottom);
		float f_dy = (fixedPixel_left_bottom + 2 * fixedPixel_bottom + fixedPixel_right_bottom) - (fixedPixel_left_top + 2 * fixedPixel_top + fixedPixel_right_top);

		float grad_m_mod = sqrtf(m_dx * m_dx + m_dy * m_dy);
		float grad_f_mod = sqrtf(f_dx * f_dx + f_dy * f_dy);

		// printf("grad_m_mod = %f, grad_f_mod = %f\n", grad_m_mod, grad_f_mod);

		if(grad_m_mod > DRRThreshold && grad_f_mod > FixedThreshold)
		{
			// int current_num = weightNum[0];
			s_num[s_idx] = 1;
			// num++;

			float cos_theta = (f_dx * m_dx + f_dy * m_dy) / (grad_f_mod * grad_m_mod);
			if(cos_theta > 1) cos_theta = 1;
			if(cos_theta < -1) cos_theta = -1;

			float w = (2 - log(fabsf(acos(cos_theta)) + 1)) / 2;
			// float current_sum = weightSum[0];
			
			s_sum[s_idx] = w;

			// printf("%d\n", *weightNum);
			// printf("w = %f\n", w);
		}
	}
	__syncthreads();

	if (s_idx == 0) {
		float block_sum = 0;
		int block_num = 0;
		for(int j = 0; j < blockDim.x * blockDim.y; ++j) {
			block_sum += s_sum[j];
			block_num += s_num[j];
		}
		// printf("block_sum = %f, block_num = %d\n", block_sum, block_num);
		atomicAdd(weightSum, block_sum);
		atomicAdd(weightNum, block_num);
	}
}

__global__ void computeMetricMedian_kernel(float *DRRGradientMap,
	float *FixedGradientMap,
	int DRRsize0,
	int DRRsize1,
	float *DRRArray,
	float *FixedArray) {

	// DRR and fixed image indeces
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	// DRR and fixed array index
	int idx = row + DRRsize0 * col;

	// printf("Thread index %i\n", DRRidx);
	// printf("row, col: %d, %d\n", row, col);

	if (DRRsize0 < idx && idx < DRRsize0 * (DRRsize1 - 1) && 
		idx % DRRsize0 != 0 && 
		idx % DRRsize0 != DRRsize0 - 1
		) { // checks if thread index is within the length of the DRR array

		// printf("index:%d\n", idx);

		float drrPixel = DRRArray[idx];
		float drrPixel_left = DRRArray[idx - 1];
		float drrPixel_right = DRRArray[idx + 1];
		float drrPixel_top = DRRArray[idx - DRRsize0];
		float drrPixel_bottom = DRRArray[idx + DRRsize0];
		float drrPixel_left_top = DRRArray[idx - DRRsize0 - 1];
		float drrPixel_right_top = DRRArray[idx - DRRsize0 + 1];
		float drrPixel_left_bottom = DRRArray[idx + DRRsize0 - 1];
		float drrPixel_right_bottom = DRRArray[idx + DRRsize0 + 1];
		float fixedPixel = FixedArray[idx];
		float fixedPixel_left = FixedArray[idx - 1];
		float fixedPixel_right = FixedArray[idx + 1];
		float fixedPixel_top = FixedArray[idx - DRRsize0];
		float fixedPixel_bottom = FixedArray[idx + DRRsize0];
		float fixedPixel_left_top = FixedArray[idx - DRRsize0 - 1];
		float fixedPixel_right_top = FixedArray[idx - DRRsize0 + 1];
		float fixedPixel_left_bottom = FixedArray[idx + DRRsize0 - 1];
		float fixedPixel_right_bottom = FixedArray[idx + DRRsize0 + 1];

		// if(idx == 599) {
		// 	printf("drrPixel:%f\n", drrPixel);
		// 	printf("drrPixel:%f\n", drrPixel_left);
		// 	printf("drrPixel:%f\n", drrPixel_right);
		// 	printf("drrPixel:%f\n", drrPixel_top);
		// 	printf("drrPixel:%f\n", drrPixel_bottom);
		// 	printf("fixedPixel:%f\n", fixedPixel);
		// 	printf("fixedPixel:%f\n", fixedPixel_left);
		// 	printf("fixedPixel:%f\n", fixedPixel_right);
		// 	printf("fixedPixel:%f\n", fixedPixel_top);
		// 	printf("fixedPixel:%f\n", fixedPixel_bottom);
		// }

		float m_dx = (drrPixel_right_top + 2 * drrPixel_right + drrPixel_right_bottom) - (drrPixel_left_top + 2 * drrPixel_left + drrPixel_left_bottom);
		float m_dy = (drrPixel_left_bottom + 2 * drrPixel_bottom + drrPixel_right_bottom) - (drrPixel_left_top + 2 * drrPixel_top + drrPixel_right_top);
		float f_dx = (fixedPixel_right_top + 2 * fixedPixel_right + fixedPixel_right_bottom) - (fixedPixel_left_top + 2 * fixedPixel_left + fixedPixel_left_bottom);
		float f_dy = (fixedPixel_left_bottom + 2 * fixedPixel_bottom + fixedPixel_right_bottom) - (fixedPixel_left_top + 2 * fixedPixel_top + fixedPixel_right_top);

		float grad_m_mod = sqrtf(m_dx * m_dx + m_dy * m_dy);
		float grad_f_mod = sqrtf(f_dx * f_dx + f_dy * f_dy);

		// printf("grad_m_mod = %f, grad_f_mod = %f\n", grad_m_mod, grad_f_mod);
		
		int col_new = int(idx / DRRsize0) - 1;
		int row_new = int(idx % DRRsize0) - 1;

		int idx_new = row_new + (DRRsize0 - 1) * col_new;

		DRRGradientMap[idx_new] = grad_m_mod;
		FixedGradientMap[idx_new] = grad_f_mod;

	}

}

__global__ void backwardProp_kernel(float *movImgArray,
	float *DRRarray,
	float *source,
	float *DestArray,
	int DRRsize0,
	int DRRsize1,
	int *MovSize,
	float *MovSpacing,
	float X0, float Y0, float Z0) {

	// DRR image indeces
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	// DRR array index
	int DRRidx = row + DRRsize0 * col;

	// moving image total size
	int totalMovSize = MovSize[0] * MovSize[1] * MovSize[2];

	if (DRRidx < DRRsize0 * DRRsize1) { // checks if thread index is within the length of the DRR array

		// --- declaration of variables for Siddon ---
		float alpha_min, alpha_max;
		float alpha_x_min, alpha_x_max, alpha_y_min, alpha_y_max, alpha_z_min, alpha_z_max;
		int i_min, i_max, j_min, j_max, k_min, k_max; // indeces corresponding to first and last intersected voxels
		float alpha_current;
		float alpha_x_next;
		float alpha_y_next;
		float alpha_z_next;
		float distance;
		int arrayIdx;
		int arrayIdx_old;
		unsigned int i_v, j_v, k_v;
		float alpha_first_pixel;
		float density_value = 0.;

		// --- define destination point based on DRR array index --- 
		float dest[3];
		get_dest(DRRidx, DestArray, dest);

		// --- source-to-destination distance --- 
		distance = sqrtf((dest[0] - source[0])*(dest[0] - source[0]) +
			(dest[1] - source[1])*(dest[1] - source[1]) +
			(dest[2] - source[2])*(dest[2] - source[2]));

		float dx = MovSpacing[0] / fabsf(dest[0] - source[0]);
		float dy = MovSpacing[1] / fabsf(dest[1] - source[1]);
		float dz = MovSpacing[2] / fabsf(dest[2] - source[2]);

		// --- find alpha_min and alpha_max
		// initialize alpha_min and alpha_max
		alpha_min = 0.;
		alpha_max = 1.;

		// X
		if (fabsf(dest[0] - source[0]) > epsilon) {

			float alpha_x0 = (X0 - source[0]) / (dest[0] - source[0]);
			float alpha_xN;
			compute_alpha_x(X0, MovSpacing[0], MovSize[0], source[0], dest[0], alpha_xN);
			alpha_x_min = fminf(alpha_x0, alpha_xN);
			alpha_x_max = fmaxf(alpha_x0, alpha_xN);
			if (alpha_x_min > alpha_min) { alpha_min = alpha_x_min; };
			if (alpha_x_max < alpha_max) { alpha_max = alpha_x_max; };
			//printf("alpha_min: %f\n", alpha_min);
			//printf("alpha_max: %f\n", alpha_max);

		}

		// Y
		if (fabsf(dest[1] - source[1]) > epsilon) {

			float alpha_y0 = (Y0 - source[1]) / (dest[1] - source[1]);
			float alpha_yN;
			compute_alpha_y(Y0, MovSpacing[1], MovSize[1], source[1], dest[1], alpha_yN);
			alpha_y_min = fminf(alpha_y0, alpha_yN);
			alpha_y_max = fmaxf(alpha_y0, alpha_yN);
			if (alpha_y_min > alpha_min) { alpha_min = alpha_y_min; };
			if (alpha_y_max < alpha_max) { alpha_max = alpha_y_max; };

		}

		// Z
		if (fabsf(dest[2] - source[2]) > epsilon) {

			float alpha_z0 = (Z0 - source[2]) / (dest[2] - source[2]);
			float alpha_zN;
			compute_alpha_z(Z0, MovSpacing[2], MovSize[2], source[2], dest[2], alpha_zN);
			alpha_z_min = fminf(alpha_z0, alpha_zN);
			alpha_z_max = fmaxf(alpha_z0, alpha_zN);
			if (alpha_z_min > alpha_min) { alpha_min = alpha_z_min; };
			if (alpha_z_max < alpha_max) { alpha_max = alpha_z_max; };

		}

		// --- initialize alpha --- 
		alpha_current = alpha_min;

		if (alpha_min < alpha_max) {

			// compute i_min, i_max and initialize alpha_x_next 
			if (dest[0] - source[0] > 0.) {

				// i_min
				if (fabsf(alpha_min - alpha_x_min) < epsilon) { i_min = 1; } // in other words: if source point is out of 3D model(x plane parallel region)
				else {
					float phi_x;
					compute_phi_x(X0, MovSpacing[0], alpha_min, source[0], dest[0], phi_x);
					i_min = ceil(phi_x);
				}

				// i_max
				if (fabsf(alpha_max - alpha_x_max) < epsilon) { i_max = MovSize[0] - 1; } // in other words: if destination point is out of 3D model(x plane parallel region)
				else {
					float phi_x;
					compute_phi_x(X0, MovSpacing[0], alpha_max, source[0], dest[0], phi_x);
					i_max = floor(phi_x);
				}

				// initialize alpha_x_next
				compute_alpha_x(X0, MovSpacing[0], i_min, source[0], dest[0], alpha_x_next);
			}
			else {

				// i_max
				if (fabsf(alpha_min - alpha_x_min) < epsilon) { i_max = MovSize[0] - 1; }
				else {
					float phi_x;
					compute_phi_x(X0, MovSpacing[0], alpha_min, source[0], dest[0], phi_x);
					i_max = floor(phi_x);
				}

				// i_min
				if (fabsf(alpha_max - alpha_x_max) < epsilon) { i_min = 1; }
				else {
					float phi_x;
					compute_phi_x(X0, MovSpacing[0], alpha_max, source[0], dest[0], phi_x);
					i_min = ceil(phi_x);
				}

				// initialize alpha_x_next
				compute_alpha_x(X0, MovSpacing[0], i_max, source[0], dest[0], alpha_x_next);
			}

			// compute j_min, j_max and initialize alpha_y_next 
			if (dest[1] - source[1] > 0.) {

				// j_min
				if (fabsf(alpha_min - alpha_y_min) < epsilon) { j_min = 1; }
				else {
					float phi_y;
					compute_phi_y(Y0, MovSpacing[1], alpha_min, source[1], dest[1], phi_y);
					//printf("phi_y: %f\n", phi_y);
					j_min = ceil(phi_y);
					//printf("j_min: %f\n", j_min);
				}

				// j_max
				if (fabsf(alpha_max - alpha_y_max) < epsilon) { j_max = MovSize[1] - 1; }
				else {
					float phi_y;
					compute_phi_y(Y0, MovSpacing[1], alpha_max, source[1], dest[1], phi_y);
					j_max = floor(phi_y);
				}

				// initialize alpha_y_next
				compute_alpha_y(Y0, MovSpacing[1], j_min, source[1], dest[1], alpha_y_next);
			}
			else {

				// j_max
				if (fabsf(alpha_min - alpha_y_min) < epsilon) { j_max = MovSize[1] - 1; }
				else {
					float phi_y;
					compute_phi_y(Y0, MovSpacing[1], alpha_min, source[1], dest[1], phi_y);
					j_max = floor(phi_y);
				}

				// j_min
				if (fabsf(alpha_max - alpha_y_max) < epsilon) { j_min = 1; }
				else {
					float phi_y;
					compute_phi_y(Y0, MovSpacing[1], alpha_max, source[1], dest[1], phi_y);
					j_min = ceil(phi_y);
				}

				// initialize alpha_y_next
				compute_alpha_y(Y0, MovSpacing[1], j_max, source[1], dest[1], alpha_y_next);
			}

			// compute k_min, k_max and initialize alpha_z_next 
			if (dest[2] - source[2] > 0.) {

				// k_min
				if (fabsf(alpha_min - alpha_z_min) < epsilon) { k_min = 1; }
				else {
					float phi_z;
					compute_phi_z(Z0, MovSpacing[2], alpha_min, source[2], dest[2], phi_z);
					k_min = ceil(phi_z);
				}

				// k_max
				if (fabsf(alpha_max - alpha_z_max) < epsilon) { k_max = MovSize[2] - 1; }
				else {
					float phi_z;
					compute_phi_z(Z0, MovSpacing[2], alpha_max, source[2], dest[2], phi_z);
					k_max = floor(phi_z);
				}

				// initialize alpha_z_next
				compute_alpha_z(Z0, MovSpacing[2], k_min, source[2], dest[2], alpha_z_next);
			}
			else {

				// k_max
				if (fabsf(alpha_min - alpha_z_min) < epsilon) { k_max = MovSize[2] - 1; }
				else {
					float phi_z;
					compute_phi_z(Z0, MovSpacing[2], alpha_min, source[2], dest[2], phi_z);
					k_max = floor(phi_z);
				}

				// k_min
				if (fabsf(alpha_max - alpha_z_max) < epsilon) { k_min = 1; }
				else {
					float phi_z;
					compute_phi_z(Z0, MovSpacing[2], alpha_max, source[2], dest[2], phi_z);
					k_min = ceil(phi_z);
				}

				// initialize alpha_z_next
				compute_alpha_z(Z0, MovSpacing[2], k_max, source[2], dest[2], alpha_z_next);
			}

			// --- initialize first intersected pixel i_v, j_v, k_v --- 
			if ((alpha_y_next < alpha_x_next) && (alpha_y_next < alpha_z_next)) {

				alpha_first_pixel = (alpha_y_next + alpha_min) / 2.;
			}
			else if (alpha_x_next < alpha_z_next) {

				alpha_first_pixel = (alpha_x_next + alpha_min) / 2.;
			}
			else {

				alpha_first_pixel = (alpha_z_next + alpha_min) / 2.;
			}


			float phi_x = 0.;
			float phi_y = 0.;
			float phi_z = 0.;
			compute_phi_x(X0, MovSpacing[0], alpha_first_pixel, source[0], dest[0], phi_x);
			i_v = floor(phi_x);
			compute_phi_y(Y0, MovSpacing[1], alpha_first_pixel, source[1], dest[1], phi_y);
			j_v = floor(phi_y);
			compute_phi_z(Z0, MovSpacing[2], alpha_first_pixel, source[2], dest[2], phi_z);
			k_v = floor(phi_z);

			// initialize array index of first intersected pixel
			arrayIdx = i_v + MovSize[0] * (j_v + MovSize[1] * k_v);
			arrayIdx_old = i_v + MovSize[0] * (j_v + MovSize[1] * k_v);

			// iterator indeces
			int stop = (i_max - i_min + 1) + (j_max - j_min + 1) + (k_max - k_min + 1);
			int iter = 0;

			//while (alpha_current < 1. && alpha_current < alpha_max) {
			while (iter < stop) {

				float l;

				// next intersection plane is y
				if ((alpha_y_next < alpha_x_next) && (alpha_y_next < alpha_z_next)) {

					l = (alpha_y_next - alpha_current);

					alpha_current = alpha_y_next;

					// update
					alpha_y_next += dy;
					j_v += (dest[1] - source[1] > 0.) ? 1 : -1;

				}

				else if (alpha_x_next < alpha_z_next) {

					// next intersection plane is x
					l = (alpha_x_next - alpha_current);

					alpha_current = alpha_x_next;

					// update
					alpha_x_next += dx;
					i_v += (dest[0] - source[0] > 0.) ? 1 : -1;


				}

				else {

					// next intersection plane is z
					l = (alpha_z_next - alpha_current);

					alpha_current = alpha_z_next;

					// update
					alpha_z_next += dz;
					k_v += (dest[2] - source[2] > 0.) ? 1 : -1;
				}

				// update array index
				update_idx(i_v, j_v, k_v, MovSize[0], MovSize[1], arrayIdx);

				if (0 < arrayIdx_old && arrayIdx_old < totalMovSize){
					
					// update density value
					// if(arrayIdx_old == 1000000){
					// 	printf("movImgArray[%d] = %f\n", arrayIdx_old, movImgArray[arrayIdx_old]);
					// 	printf("add = %f\n", DRRarray[DRRidx] * l * distance);
					// 	printf("DRRidx = %d\n", DRRidx);
					// 	printf("row, col: %d, %d\n", row, col);
					// 	atomicAdd(&movImgArray[arrayIdx_old], DRRarray[DRRidx] * l * distance);
					// 	printf("movImgArray[%d] = %f\n", arrayIdx_old, movImgArray[arrayIdx_old]);
					// }
					
					atomicAdd(&movImgArray[arrayIdx_old], DRRarray[DRRidx] * l * distance);
					

				}

				// update arrayIdx
				arrayIdx_old = arrayIdx;

				// update iter
				iter += 1;

			}

		}

	}

}

/**
*
* Deafult constructor
*
**/
SiddonGpu::SiddonGpu() { }

/**
*
* Overloaded constructor loads the CT scan (together with size and spacing) onto GPU memory
*
**/
SiddonGpu::SiddonGpu(int *NumThreadsPerBlock,
	float *movImgArray,
	int *MovSize,
	float *MovSpacing,
	float X0, float Y0, float Z0,
	int *DRRSize) {

	// ---- Allocate variable members ---- 
	m_NumThreadsPerBlock[0] = NumThreadsPerBlock[0];
	m_NumThreadsPerBlock[1] = NumThreadsPerBlock[1];
	m_NumThreadsPerBlock[2] = NumThreadsPerBlock[2];

	//m_MovSize[0] = MovSize[0];
	//m_MovSize[1] = MovSize[1];
	//m_MovSize[2] = MovSize[2];

	m_X0 = X0;
	m_Y0 = Y0;
	m_Z0 = Z0;

	m_DRRsize[0] = DRRSize[0];
	m_DRRsize[1] = DRRSize[1];
	m_DRRsize[2] = DRRSize[2];

	m_DRRsize0 = DRRSize[0];
	m_DRRsize1 = DRRSize[1];

	m_movImgMemSize = MovSize[0] * MovSize[1] * MovSize[2] * sizeof(float);
	m_DestMemSize = (DRRSize[0] * DRRSize[1] * DRRSize[2] * 3) * sizeof(float);
	m_DrrMemSize = (DRRSize[0] * DRRSize[1] * DRRSize[2]) * sizeof(float); // memory for each output drr

	// allocate space for device copies
	hipMalloc((void**)&m_d_movImgArray, m_movImgMemSize);
	hipMalloc((void**)&m_d_MovSize, 3 * sizeof(int));
	hipMalloc((void**)&m_d_MovSpacing, 3 * sizeof(float));

	// Copy arrays related to the moving image onto device array
	hipMemcpy(m_d_movImgArray, movImgArray, m_movImgMemSize, hipMemcpyHostToDevice);
	hipMemcpy(m_d_MovSize, MovSize, 3 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(m_d_MovSpacing, MovSpacing, 3 * sizeof(float), hipMemcpyHostToDevice);

	//std::cout << "Siddon object Initialization: GPU memory prepared \n" << std::endl;
	//printf("ctor %p\n", this); // in constructors

}

/**
*
* Destructor clears everything left from the GPU memory
*
**/
SiddonGpu::~SiddonGpu() {

	hipFree(m_d_movImgArray);
	hipFree(m_d_MovSize);
	hipFree(m_d_MovSpacing);
	//hipFree(d_drr_array);
	//std::cout << "Siddon object destruction: GPU memory cleared \n" << std::endl;
	//printf("dtor %p\n", this); // in destructor

}

/**
*-The function generate DRR must be called with the following variables :
*
* @param source : array of(transformed) source physical coordinates
* @param DestArray : C - ordered 1D array of physical coordinates relative to the(transformed) output DRR image.
* @param drrArray : output, 1D array for output values of projected CT densities
*
**/
void SiddonGpu::generateDRR(float *source,
							float *DestArray,
							float *drrArray) {

	hipError_t ierrAsync;
	hipError_t ierrSync;

	// declare pointer to device memory for output DRR array
	float *d_DestArray;
	float *d_source;
	// float *d_drr_array;

	// allocate space on device
	hipMalloc((void**)&d_drr_array, m_DrrMemSize);
	hipMalloc((void**)&d_source, 3 * sizeof(float));
	hipMalloc((void**)&d_DestArray, m_DestMemSize);

	// Copy source and destination to device
	hipMemcpy(d_DestArray, DestArray, m_DestMemSize, hipMemcpyHostToDevice);
	hipMemcpy(d_source, source, 3 * sizeof(float), hipMemcpyHostToDevice);

	// printf("cudaMalloc1OK!\n");

	//std::cout << "DRR generation: GPU memory prepared \n" << std::endl;

	// determine number of required blocks
	dim3 threads_per_block(m_NumThreadsPerBlock[0], m_NumThreadsPerBlock[1], 1);
	dim3 number_of_blocks((m_DRRsize[0] / threads_per_block.x) + 1, (m_DRRsize[1] / threads_per_block.y) + 1, 1);

	//// Query GPU device
	//hipDeviceProp_t prop;
	//hipGetDeviceProperties(&prop, 0);
	//std::cout << "Max threads per block " << prop.maxThreadsPerBlock << std::endl;
	//hipGetDeviceProperties(&prop, 0);
	//if (threads_per_block.x * threads_per_block.y * threads_per_block.z > prop.maxThreadsPerBlock) {
	//	printf("Too many threads per block ... exiting\n");
	//	goto cleanup;
	//}
	//if (threads_per_block.x > prop.maxThreadsDim[0]) {
	//	printf("Too many threads in x-direction ... exiting\n");
	//	goto cleanup;
	//}
	//if (threads_per_block.y > prop.maxThreadsDim[1]) {
	//	printf("Too many threads in y-direction ... exiting\n");
	//	goto cleanup;
	//}
	//if (threads_per_block.z > prop.maxThreadsDim[2]) {
	//	printf("Too many threads in z-direction ... exiting\n");
	//	goto cleanup;
	//}

	// launch kernel
	generateDRR_kernel << <number_of_blocks, threads_per_block >> >(d_drr_array,
															 d_source,
															 d_DestArray,
															 m_DRRsize0,
															 m_DRRsize1,
															 m_d_movImgArray,
															 m_d_MovSize,
															 m_d_MovSpacing,
															 m_X0, m_Y0, m_Z0);


	// Check for errors in Kernel launch
	ierrSync = hipGetLastError();
	ierrAsync = hipDeviceSynchronize(); // Wait for the GPU to finish
	if (ierrSync != hipSuccess) { 
		printf("Cuda Sync error: %s\n", hipGetErrorString(ierrSync));
		//goto cleanup; 
	}
	if (ierrAsync != hipSuccess) { 
		printf("Cuda Async error: %s\n", hipGetErrorString(ierrAsync)); 
		//goto cleanup;
	}

	// printf("kernel1finished!\n");

	// Copy result to host array
	hipMemcpy(drrArray, d_drr_array, m_DrrMemSize, hipMemcpyDeviceToHost);

	// printf("drrArray[599]:%f\n", drrArray[599]);

	// Clean up device DRR array
cleanup:
	hipFree(d_source);
	hipFree(d_DestArray);
	//std::cout << "DRR generation: GPU memory cleared \n" << std::endl;

	return;

}

void SiddonGpu::computeMetric(float *fixedArray,
	float drrThreshold,
	float fixedThreshold,
	int lowNum,
	float *weightSum,
	int *weightNum,
	float *metricValue) {
	
	// printf("start initialising!\n");

	hipError_t ierrAsync;
	hipError_t ierrSync;

	// declare pointer to device memory for metric
	float *d_fixed_array;
	float *d_weight_sum;
	int *d_weight_num;
	float d_drr_threshold = drrThreshold;
	float d_fixed_threshold = fixedThreshold;

	// printf("initialised!\n");

	// allocate space on device
	// hipMalloc((void**)&d_metric_value, sizeof(float));
	hipMalloc((void**)&d_fixed_array, m_DrrMemSize);
	hipMalloc((void**)&d_weight_sum, sizeof(float));
	hipMalloc((void**)&d_weight_num, sizeof(int));

	// printf("cudaMalloc2OK!\n");

	// Copy source and destination to device
	hipMemcpy(d_fixed_array, fixedArray, m_DrrMemSize, hipMemcpyHostToDevice);
	hipMemset(d_weight_sum, 0, sizeof(float));
	hipMemset(d_weight_num, 0, sizeof(int));

	// hipMemcpy(d_test_drr, d_drr_array, m_DrrMemSize, hipMemcpyDeviceToHost);

	// printf("cudacopy2OK!\n");
	// printf("fixedArray[599]:%f\n", fixedArray[599]);

	// determine number of required blocks
	dim3 threads_per_block(m_NumThreadsPerBlock[0], m_NumThreadsPerBlock[1], 1);
	dim3 number_of_blocks((m_DRRsize[0] / threads_per_block.x) + 1, (m_DRRsize[1] / threads_per_block.y) + 1, 1);

	// launch kernel
	computeMetric_kernel << <number_of_blocks, threads_per_block >> >(d_weight_sum, 
																	d_weight_num, 
																	m_DRRsize0,
																	m_DRRsize1,
																	d_drr_array, 
																	d_fixed_threshold,
																	d_drr_threshold,
																	d_fixed_array);

	// Check for errors in Kernel launch
	ierrSync = hipGetLastError();
	ierrAsync = hipDeviceSynchronize(); // Wait for the GPU to finish
	if (ierrSync != hipSuccess) { 
		printf("Cuda Sync error2: %s\n", hipGetErrorString(ierrSync));
		//goto cleanup; 
	}
	if (ierrAsync != hipSuccess) { 
		printf("Cuda Async error2: %s\n", hipGetErrorString(ierrAsync)); 
		//goto cleanup;
	}

	// Copy result to host array
	hipMemcpy(weightSum, d_weight_sum, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(weightNum, d_weight_num, sizeof(int), hipMemcpyDeviceToHost);

	// Compute metric value
	if(*weightNum > lowNum) {
		*metricValue = (1 - *weightSum / *weightNum);
	}
	else {
		*metricValue = (1 - *weightSum / lowNum);
	}
	printf("weightnum:%d\n", *weightNum);

	// Clean up device DRR array
cleanup:
	hipFree(d_weight_sum);
	hipFree(d_weight_num);
	hipFree(d_fixed_array);
	hipFree(d_drr_array);

	return;
}

void SiddonGpu::backwardProp(float *source,
								float *DestArray,
								float *inputGradArray, 
								float *outputGradArray){
	hipError_t ierrAsync;
	hipError_t ierrSync;

	// declare pointer to device memory for output DRR array
	float *d_DestArray;
	float *d_source;
	float *d_inputGradArray;

	// allocate space on device
	hipMalloc((void**)&d_inputGradArray, m_DrrMemSize);
	hipMalloc((void**)&d_source, 3 * sizeof(float));
	hipMalloc((void**)&d_DestArray, m_DestMemSize);

	// Copy source and destination to device
	hipMemcpy(d_inputGradArray, inputGradArray, m_DrrMemSize, hipMemcpyHostToDevice); // Copy drr gradient array to device
	hipMemcpy(d_DestArray, DestArray, m_DestMemSize, hipMemcpyHostToDevice); // Copy drr destination array to device
	hipMemcpy(d_source, source, 3 * sizeof(float), hipMemcpyHostToDevice); // Copy source point array to device

	// printf("cudaMalloc1OK!\n");

	// determine number of required blocks
	dim3 threads_per_block(m_NumThreadsPerBlock[0], m_NumThreadsPerBlock[1], 1);
	dim3 number_of_blocks((m_DRRsize[0] / threads_per_block.x) + 1, (m_DRRsize[1] / threads_per_block.y) + 1, 1);

	// launch kernel
	backwardProp_kernel << <number_of_blocks, threads_per_block >> >(m_d_movImgArray,
															 d_inputGradArray,
															 d_source,
															 d_DestArray,
															 m_DRRsize0,
															 m_DRRsize1,
															 m_d_MovSize,
															 m_d_MovSpacing,
															 m_X0, m_Y0, m_Z0);


	// Check for errors in Kernel launch
	ierrSync = hipGetLastError();
	ierrAsync = hipDeviceSynchronize(); // Wait for the GPU to finish
	if (ierrSync != hipSuccess) { 
		printf("Cuda Sync error3: %s\n", hipGetErrorString(ierrSync));
		//goto cleanup; 
	}
	if (ierrAsync != hipSuccess) { 
		printf("Cuda Async error3: %s\n", hipGetErrorString(ierrAsync)); 
		//goto cleanup;
	}

	// printf("kernel1finished!\n");

	// Copy result to host array
	hipMemcpy(outputGradArray, m_d_movImgArray, m_movImgMemSize, hipMemcpyDeviceToHost);

	// printf("drrArray[599]:%f\n", drrArray[599]);

	// Clean up device DRR array
cleanup:
	hipFree(d_source);
	hipFree(d_DestArray);
	hipFree(d_inputGradArray);
	//std::cout << "DRR generation: GPU memory cleared \n" << std::endl;

	return;
}

void SiddonGpu::computeMetricMedian(float *fixedArray,
	float *drrGradientMap,
	float *fixedGradientMap) {
	
	// printf("start initialising!\n");

	hipError_t ierrAsync;
	hipError_t ierrSync;

	// declare pointer to device memory for metric
	float *d_fixed_array;
	float *d_drrGradientMap;
	float *d_fixedGradientMap;

	// printf("initialised!\n");

	// allocate space on device
	// hipMalloc((void**)&d_metric_value, sizeof(float));
	hipMalloc((void**)&d_fixed_array, m_DrrMemSize);
	hipMalloc((void**)&d_drrGradientMap, ((m_DRRsize[0] - 1) * (m_DRRsize[1] - 1) * m_DRRsize[2]) * sizeof(float));
	hipMalloc((void**)&d_fixedGradientMap, ((m_DRRsize[0] - 1) * (m_DRRsize[1] - 1) * m_DRRsize[2]) * sizeof(float));

	// printf("cudaMalloc2OK!\n");

	// Copy source and destination to device
	hipMemcpy(d_fixed_array, fixedArray, m_DrrMemSize, hipMemcpyHostToDevice);

	// hipMemcpy(d_test_drr, d_drr_array, m_DrrMemSize, hipMemcpyDeviceToHost);

	// printf("cudacopy2OK!\n");
	// printf("fixedArray[599]:%f\n", fixedArray[599]);

	// determine number of required blocks
	dim3 threads_per_block(m_NumThreadsPerBlock[0], m_NumThreadsPerBlock[1], 1);
	dim3 number_of_blocks((m_DRRsize[0] / threads_per_block.x) + 1, (m_DRRsize[1] / threads_per_block.y) + 1, 1);

	// launch kernel
	computeMetricMedian_kernel << <number_of_blocks, threads_per_block >> >(d_drrGradientMap,
																	d_fixedGradientMap,
																	m_DRRsize0,
																	m_DRRsize1,
																	d_drr_array,
																	d_fixed_array);

	// Check for errors in Kernel launch
	ierrSync = hipGetLastError();
	ierrAsync = hipDeviceSynchronize(); // Wait for the GPU to finish
	if (ierrSync != hipSuccess) { 
		printf("Cuda Sync error4: %s\n", hipGetErrorString(ierrSync));
		//goto cleanup; 
	}
	if (ierrAsync != hipSuccess) { 
		printf("Cuda Async error4: %s\n", hipGetErrorString(ierrAsync)); 
		//goto cleanup;
	}

	// Copy result to host array
	hipMemcpy(drrGradientMap, d_drrGradientMap, ((m_DRRsize[0] - 1) * (m_DRRsize[1] - 1) * m_DRRsize[2]) * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(fixedGradientMap, d_fixedGradientMap, ((m_DRRsize[0] - 1) * (m_DRRsize[1] - 1) * m_DRRsize[2]) * sizeof(float), hipMemcpyDeviceToHost);

	// Clean up device DRR array
cleanup:
	hipFree(d_fixed_array);
	hipFree(d_drrGradientMap);
	hipFree(d_fixedGradientMap);

	return;
}
